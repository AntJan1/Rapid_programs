#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <string.h>



__global__ void vfun(short *vectrzy, long long n, long long *gpup, long long *xgpu,long long it,long long g)
{
    long long res; long long resx;long long rowx;long long col;long long row; long long size=(long long)pow(2,(n-1));
    long long i=blockIdx.x * blockDim.x + threadIdx.x+size/pow(2,it)*g;
    long long ig=blockIdx.x * blockDim.x + threadIdx.x;
    *(gpup+ig)=0;
    for(long long j=0;j<size;j++)
    {
        res=1;
        resx=0;
        col=(i*size+j)/size;
        row=(i*size+j)-size*col;
        rowx=j;
        for(long long k=0;k<(n-1)/2;k++)
        {
            res*=*(vectrzy+row%4+(col%4)*4);
            row/=4;
            col/=4;
        }
        for(long long k=0;k<(n-1)/2;k++)
        {
//             printf("%d",resx);
            resx+=*(xgpu+rowx%4);
//              printf("%d\n",resx);
            rowx/=4;
        }
        resx++;
        if(resx<0)
            resx*=-1;
     *(gpup+ig)+=res*resx;
    }
//     printf("%lld\n",*(gpup+i));
}

int main(int argv,char *argc[])
{
    FILE *wynik;
    wynik=fopen("eigenvalues.txt","w+");
    clock_t start = clock();
    long long n;
    long long *gpux;long long *gpup; short *vec3gpu;
    n=atoi(argc[1]);

    unsigned long long sn=pow(2,n-1);
    long long it=0;
    for(;sn>pow(2,28);)
    {
     sn/=2;
     it++;
    }
     unsigned long long size=sn*sizeof(long long);

         long long threadsperblock = (long long)pow(2,(n-1)/2);
    if(threadsperblock>1024)
        threadsperblock=1024;
    long long blockspergrid=(sn + threadsperblock -1) / threadsperblock;

    long long *p=(long long *)malloc(size);
    short *vec=(short *)malloc(16*sizeof(short));
    hipMalloc((void **)&gpux, sizeof(long long)*4);
    hipMalloc((void **)&gpup,size);
    hipMalloc((void **)&vec3gpu,16*sizeof(short));

// Calculating eigenvalues
        *vec=1;*(vec+1)=1;*(vec+2)=1;*(vec+3)=1;*(vec+4)=1;*(vec+5)=1;*(vec+6)=-1;*(vec+7)=-1;*(vec+8)=1;*(vec+9)=-1;*(vec+10)=-1;*(vec+11)=1;*(vec+12)=1;*(vec+13)=-1;*(vec+14)=1;*(vec+15)=-1;
        *p=2;*(p+1)=0;*(p+2)=-2;*(p+3)=0;
        hipMemcpy(gpux, p, sizeof(long long)*4, hipMemcpyHostToDevice);
        hipMemcpy(vec3gpu,vec,16*sizeof(short),hipMemcpyHostToDevice);

        for(long long g=0;g<pow(2,it);g++)
        {
            vfun<<<blockspergrid,threadsperblock>>>(vec3gpu, n, gpup, gpux,it,g);
            hipDeviceSynchronize();
            hipMemcpy(p,gpup,size,hipMemcpyDeviceToHost);

            for(long long i=0;i<sn;i++)
            {
                fprintf(wynik,"%lld\n",*(p+i));
                if(*(p+i)==0){
                printf("FATAL ERROR ");break;}
            }
        }
// clock
      clock_t end = clock();
      float t = (float)(end-start)/CLOCKS_PER_SEC;
      printf("%f\n",t);

//free memory
     hipFree(gpux);
     hipFree(gpup);
     hipFree(vec3gpu);
     free(vec);
     free(p);
     fclose(wynik);
     hipDeviceReset();
    return 0;
}
